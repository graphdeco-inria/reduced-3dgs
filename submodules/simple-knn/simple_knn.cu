#include "hip/hip_runtime.h"
/*
 * Copyright (C) 2023, Inria
 * GRAPHDECO research group, https://team.inria.fr/graphdeco
 * All rights reserved.
 *
 * This software is free for non-commercial, research and evaluation use 
 * under the terms of the LICENSE.md file.
 *
 * For inquiries contact  george.drettakis@inria.fr
 */

#define BOX_SIZE 1024
#define BOX_SIZE2 128

#include "hip/hip_runtime.h"
#include ""
#include "simple_knn.h"
#include <hipcub/hipcub.hpp>
#include <cub/device/device_radix_sort.cuh>
#include <vector>
#include <hip/hip_runtime_api.h>
#include <thrust/device_vector.h>
#include <thrust/sequence.h>
#define __HIPCC__
#include <hip/hip_cooperative_groups.h>
#include <cooperative_groups/reduce.h>

namespace cg = cooperative_groups;

struct CustomMin
{
	__device__ __forceinline__
		float3 operator()(const float3& a, const float3& b) const {
		return { min(a.x, b.x), min(a.y, b.y), min(a.z, b.z) };
	}
};

struct CustomMax
{
	__device__ __forceinline__
		float3 operator()(const float3& a, const float3& b) const {
		return { max(a.x, b.x), max(a.y, b.y), max(a.z, b.z) };
	}
};

__host__ __device__ uint32_t prepMorton(uint32_t x)
{
	x = (x | (x << 16)) & 0x030000FF;
	x = (x | (x << 8)) & 0x0300F00F;
	x = (x | (x << 4)) & 0x030C30C3;
	x = (x | (x << 2)) & 0x09249249;
	return x;
}

__host__ __device__ uint32_t coord2Morton(float3 coord, float3 minn, float3 maxx)
{
	uint32_t x = prepMorton(((coord.x - minn.x) / (maxx.x - minn.x)) * ((1 << 10) - 1));
	uint32_t y = prepMorton(((coord.y - minn.y) / (maxx.y - minn.y)) * ((1 << 10) - 1));
	uint32_t z = prepMorton(((coord.z - minn.z) / (maxx.z - minn.z)) * ((1 << 10) - 1));

	return x | (y << 1) | (z << 2);
}

__global__ void coord2Morton(int P, const float3* points, float3 minn, float3 maxx, uint32_t* codes)
{
	auto idx = cg::this_grid().thread_rank();
	if (idx >= P)
		return;

	codes[idx] = coord2Morton(points[idx], minn, maxx);
}

struct MinMax
{
	float3 minn;
	float3 maxx;
};

template <uint32_t B>
__global__ void boxMinMax(uint32_t P, float3* points, uint32_t* indices, MinMax* boxes)
{
	auto idx = cg::this_grid().thread_rank();

	MinMax me;
	if (idx < P)
	{
		me.minn = points[indices[idx]];
		me.maxx = points[indices[idx]];
	}
	else
	{
		me.minn = { FLT_MAX, FLT_MAX, FLT_MAX };
		me.maxx = { -FLT_MAX,-FLT_MAX,-FLT_MAX };
	}

	__shared__ MinMax redResult[BOX_SIZE];

	for (int off = BOX_SIZE / 2; off >= 1; off /= 2)
	{
		if (threadIdx.x < 2 * off)
			redResult[threadIdx.x] = me;
		__syncthreads();

		if (threadIdx.x < off)
		{
			MinMax other = redResult[threadIdx.x + off];
			me.minn.x = min(me.minn.x, other.minn.x);
			me.minn.y = min(me.minn.y, other.minn.y);
			me.minn.z = min(me.minn.z, other.minn.z);
			me.maxx.x = max(me.maxx.x, other.maxx.x);
			me.maxx.y = max(me.maxx.y, other.maxx.y);
			me.maxx.z = max(me.maxx.z, other.maxx.z);
		}
		__syncthreads();
	}

	if (threadIdx.x == 0)
		boxes[blockIdx.x] = me;
}

__device__ __host__ float distBoxPoint(const MinMax& box, const float3& p)
{
	float3 diff = { 0, 0, 0 };
	if (p.x < box.minn.x || p.x > box.maxx.x)
		diff.x = min(abs(p.x - box.minn.x), abs(p.x - box.maxx.x));
	if (p.y < box.minn.y || p.y > box.maxx.y)
		diff.y = min(abs(p.y - box.minn.y), abs(p.y - box.maxx.y));
	if (p.z < box.minn.z || p.z > box.maxx.z)
		diff.z = min(abs(p.z - box.minn.z), abs(p.z - box.maxx.z));
	return diff.x * diff.x + diff.y * diff.y + diff.z * diff.z;
}

template<int K>
__device__ void updateKBest(const float3& ref, const float3& point, float* knn)
{
	float3 d = { point.x - ref.x, point.y - ref.y, point.z - ref.z };
	float dist = d.x * d.x + d.y * d.y + d.z * d.z;
	for (int j = 0; j < K; j++)
	{
		if (knn[j] > dist)
		{
			float t = knn[j];
			knn[j] = dist;
			dist = t;
		}
	}
}

__global__ void boxMeanDist(uint32_t P, float3* points, uint32_t* indices, MinMax* boxes, float* dists)
{
	int idx = cg::this_grid().thread_rank();
	if (idx >= P)
		return;

	float3 point = points[indices[idx]];
	float best[3] = { FLT_MAX, FLT_MAX, FLT_MAX };

	for (int i = max(0, idx - 3); i <= min(P - 1, idx + 3); i++)
	{
		if (i == idx)
			continue;
		updateKBest<3>(point, points[indices[i]], best);
	}

	float reject = best[2];
	best[0] = FLT_MAX;
	best[1] = FLT_MAX;
	best[2] = FLT_MAX;

	for (int b = 0; b < (P + BOX_SIZE - 1) / BOX_SIZE; b++)
	{
		MinMax box = boxes[b];
		float dist = distBoxPoint(box, point);
		if (dist > reject || dist > best[2])
			continue;

		for (int i = b * BOX_SIZE; i < min(P, (b + 1) * BOX_SIZE); i++)
		{
			if (i == idx)
				continue;
			updateKBest<3>(point, points[indices[i]], best);
		}
	}
	dists[indices[idx]] = (best[0] + best[1] + best[2]) / 3.0f;
}

void SimpleKNN::knn(int P, float3* points, float* meanDists)
{
	float3* result;
	hipMalloc(&result, sizeof(float3));
	size_t temp_storage_bytes;

	float3 init = { 0, 0, 0 }, minn, maxx;

	hipcub::DeviceReduce::Reduce(nullptr, temp_storage_bytes, points, result, P, CustomMin(), init);
	thrust::device_vector<char> temp_storage(temp_storage_bytes);

	hipcub::DeviceReduce::Reduce(temp_storage.data().get(), temp_storage_bytes, points, result, P, CustomMin(), init);
	hipMemcpy(&minn, result, sizeof(float3), hipMemcpyDeviceToHost);

	hipcub::DeviceReduce::Reduce(temp_storage.data().get(), temp_storage_bytes, points, result, P, CustomMax(), init);
	hipMemcpy(&maxx, result, sizeof(float3), hipMemcpyDeviceToHost);

	thrust::device_vector<uint32_t> morton(P);
	thrust::device_vector<uint32_t> morton_sorted(P);
	coord2Morton << <(P + 255) / 256, 256 >> > (P, points, minn, maxx, morton.data().get());

	thrust::device_vector<uint32_t> indices(P);
	thrust::sequence(indices.begin(), indices.end());
	thrust::device_vector<uint32_t> indices_sorted(P);

	hipcub::DeviceRadixSort::SortPairs(nullptr, temp_storage_bytes, morton.data().get(), morton_sorted.data().get(), indices.data().get(), indices_sorted.data().get(), P);
	temp_storage.resize(temp_storage_bytes);

	hipcub::DeviceRadixSort::SortPairs(temp_storage.data().get(), temp_storage_bytes, morton.data().get(), morton_sorted.data().get(), indices.data().get(), indices_sorted.data().get(), P);

	uint32_t num_boxes = (P + BOX_SIZE - 1) / BOX_SIZE;
	thrust::device_vector<MinMax> boxes(num_boxes);
	boxMinMax<BOX_SIZE> << <num_boxes, BOX_SIZE >> > (P, points, indices_sorted.data().get(), boxes.data().get());
	boxMeanDist << <num_boxes, BOX_SIZE >> > (P, points, indices_sorted.data().get(), boxes.data().get(), meanDists);

	hipFree(result);
}


__device__ void updateKBest(int K, int index, const float3& ref, const float3& point, float* knn, int* indices)
{
	float3 d = { point.x - ref.x, point.y - ref.y, point.z - ref.z };
	float dist = d.x * d.x + d.y * d.y + d.z * d.z;
	int ind = index;

	for (int j = 0; j < K; j++)
	{
		if (dist < knn[j])
		{
			float t = knn[j];
			int _i = indices[j];
			knn[j] = dist;
			indices[j] = ind;
			dist = t;
			ind = _i;
		}
	}
}

__global__ void boxKnn(int K, uint32_t P, float3* points, uint32_t* indices, MinMax* boxes, float* dists, int* index_space)
{
	int idx = cg::this_grid().thread_rank();
	if (idx >= P)
		return;

	float3 point = points[indices[idx]];
	
	float* best = dists + indices[idx] * K;
	int* best_ind = index_space + indices[idx] * K;
	for(int i = 0; i < K; i++)
		best[i] = FLT_MAX;

	for (int i = max(0, idx - K); i <= min(P - 1, idx + K); i++)
	{
		if (i == idx)
			continue;
		updateKBest(K, indices[i], point, points[indices[i]], best, best_ind);
	}

	float reject = best[K-1];
	for(int i = 0; i < K; i++)
		best[i] = FLT_MAX;

	for (int b = 0; b < (P + BOX_SIZE - 1) / BOX_SIZE; b++)
	{
		MinMax box = boxes[b];
		float dist = distBoxPoint(box, point);
		if (dist > reject || dist > best[K-1])
			continue;

		for (int i = b * BOX_SIZE; i < min(P, (b + 1) * BOX_SIZE); i++)
		{
			if (i == idx)
				continue;
			updateKBest(K, indices[i], point, points[indices[i]], best, best_ind);
		}
	}
}

void SimpleKNN::knn_index(int K, int P, float3* points, float* dists, int* index_space)
{
	float3* result;
	hipMalloc(&result, sizeof(float3));
	size_t temp_storage_bytes;

	float3 init = { 0, 0, 0 }, minn, maxx;

	hipcub::DeviceReduce::Reduce(nullptr, temp_storage_bytes, points, result, P, CustomMin(), init);
	thrust::device_vector<char> temp_storage(temp_storage_bytes);

	hipcub::DeviceReduce::Reduce(temp_storage.data().get(), temp_storage_bytes, points, result, P, CustomMin(), init);
	hipMemcpy(&minn, result, sizeof(float3), hipMemcpyDeviceToHost);

	hipcub::DeviceReduce::Reduce(temp_storage.data().get(), temp_storage_bytes, points, result, P, CustomMax(), init);
	hipMemcpy(&maxx, result, sizeof(float3), hipMemcpyDeviceToHost);

	thrust::device_vector<uint32_t> morton(P);
	thrust::device_vector<uint32_t> morton_sorted(P);
	coord2Morton << <(P + 255) / 256, 256 >> > (P, points, minn, maxx, morton.data().get());

	thrust::device_vector<uint32_t> indices(P);
	thrust::sequence(indices.begin(), indices.end());
	thrust::device_vector<uint32_t> indices_sorted(P);

	hipcub::DeviceRadixSort::SortPairs(nullptr, temp_storage_bytes, morton.data().get(), morton_sorted.data().get(), indices.data().get(), indices_sorted.data().get(), P);
	temp_storage.resize(temp_storage_bytes);

	hipcub::DeviceRadixSort::SortPairs(temp_storage.data().get(), temp_storage_bytes, morton.data().get(), morton_sorted.data().get(), indices.data().get(), indices_sorted.data().get(), P);

	// hipEvent_t ev1, ev2, ev3;
	// hipEventCreate(&ev1);
	// hipEventCreate(&ev2);
	// hipEventCreate(&ev3);

	uint32_t num_boxes = (P + BOX_SIZE - 1) / BOX_SIZE;
	thrust::device_vector<MinMax> boxes(num_boxes);
	// hipEventRecord(ev1);
	boxMinMax<BOX_SIZE> << <num_boxes, BOX_SIZE >> > (P, points, indices_sorted.data().get(), boxes.data().get());
	// hipEventRecord(ev2);
	boxKnn << <num_boxes, BOX_SIZE >> > (K, P, points, indices_sorted.data().get(), boxes.data().get(), dists, index_space);
	// hipEventRecord(ev3);

	// hipEventSynchronize(ev3);
	// float ms1, ms2;
	// hipEventElapsedTime(&ms1, ev1, ev2);
	// hipEventElapsedTime(&ms2, ev2, ev3);

	// std::cout << "First part: " << ms1 << std::endl;
	// std::cout << "Second part: " << ms2 << std::endl;

	hipFree(result);
}

__device__ float get4FromK(
	int K,
	float4& dist4,
	int4& ind4,
	float* __restrict__ knn,
	int* __restrict__ indices)
{
	dist4 = { FLT_MAX, FLT_MAX, FLT_MAX, FLT_MAX };
	ind4 = { -1, -1, -1, -1 };
	for (int j = 0; j < K; j++)
	{
		float v = knn[j];
		int ind = indices[j];
		float w = v;
		int indx = ind;
		if (v < dist4.w)
		{
			if (v < dist4.z)
			{
				if (v < dist4.y)
				{
					if (v < dist4.x)
					{
						w = dist4.x;
						indx = ind4.x;
						dist4.x = v;
						ind4.x = ind;
						v = w;
						ind = indx;
					}
					w = dist4.y;
					indx = ind4.y;
					dist4.y = v;
					ind4.y = ind;
					v = w;
					ind = indx;
				}
				w = dist4.z;
				indx = ind4.z;
				dist4.z = v;
				ind4.z = ind;
				v = w;
				ind = indx;
			}
			dist4.w = v;
			ind4.w = ind;
		}
	}
}

__device__ void updateKBest2(
	float& reject,
	int K,
	int index,
	const float3& ref,
	const float3& point,
	float* __restrict__ knn,
	int* __restrict__ indices)
{
	float3 d = { point.x - ref.x, point.y - ref.y, point.z - ref.z };
	float dist = d.x * d.x + d.y * d.y + d.z * d.z;
	if (dist >= reject)
		return;

	float test_reject = dist;
	int maxint = -1;
	for (int j = 0; j < K; j++)
	{
		if (test_reject < knn[j])
		{
			test_reject = knn[j];
			maxint = j;
		}
	}
	if (maxint != -1)
	{
		knn[maxint] = dist;
		indices[maxint] = index;
	}
	reject = min(reject, test_reject);
}

__global__ void boxKnn2(
	int K,
	uint32_t P,
	const float3* __restrict__ points,
	const uint32_t* __restrict__ indices,
	const MinMax* __restrict__ boxes,
	float* __restrict__ dists,
	int* __restrict__ index_space)
{
	int idx = cg::this_grid().thread_rank();
	if (idx >= P)
		return;

	const float3 point = points[indices[idx]];

	float* best = dists + indices[idx] * K;
	int* best_ind = index_space + indices[idx] * K;
	for (int i = 0; i < K; i++)
		best[i] = FLT_MAX;

	float reject = FLT_MAX;
	int b = idx / BOX_SIZE2;
	int lo = b, hi = b;

	const int num_boxes = (P + BOX_SIZE2 - 1) / BOX_SIZE2;

	for (int iter = 0; iter < num_boxes; iter++)
	{
		MinMax box = boxes[b];
		float dist = distBoxPoint(box, point);
		if (dist < reject)
		{
			for (int i = b * BOX_SIZE2; i < min(P, (b + 1) * BOX_SIZE2); i++)
			{
				if (i == idx)
					continue;
				const int other_idx = indices[i];
				updateKBest2(reject, K, other_idx, point, points[other_idx], best, best_ind);
			}
		}
		bool odd = iter & 1;
		b = (odd && hi == num_boxes - 1) || (!odd && lo > 0) ? --lo : ++hi;
	}
}

void SimpleKNN::knn_index2(int K, int P, float3* points, float* dists, int* index_space)
{
	float3* result;
	hipMalloc(&result, sizeof(float3));
	size_t temp_storage_bytes;

	float3 init = { 0, 0, 0 }, minn, maxx;

	hipcub::DeviceReduce::Reduce(nullptr, temp_storage_bytes, points, result, P, CustomMin(), init);
	thrust::device_vector<char> temp_storage(temp_storage_bytes);

	hipcub::DeviceReduce::Reduce(temp_storage.data().get(), temp_storage_bytes, points, result, P, CustomMin(), init);
	hipMemcpy(&minn, result, sizeof(float3), hipMemcpyDeviceToHost);

	hipcub::DeviceReduce::Reduce(temp_storage.data().get(), temp_storage_bytes, points, result, P, CustomMax(), init);
	hipMemcpy(&maxx, result, sizeof(float3), hipMemcpyDeviceToHost);

	thrust::device_vector<uint32_t> morton(P);
	thrust::device_vector<uint32_t> morton_sorted(P);
	coord2Morton << <(P + 255) / 256, 256 >> > (P, points, minn, maxx, morton.data().get());

	thrust::device_vector<uint32_t> indices(P);
	thrust::sequence(indices.begin(), indices.end());
	thrust::device_vector<uint32_t> indices_sorted(P);

	hipcub::DeviceRadixSort::SortPairs(nullptr, temp_storage_bytes, morton.data().get(), morton_sorted.data().get(), indices.data().get(), indices_sorted.data().get(), P);
	temp_storage.resize(temp_storage_bytes);

	hipcub::DeviceRadixSort::SortPairs(temp_storage.data().get(), temp_storage_bytes, morton.data().get(), morton_sorted.data().get(), indices.data().get(), indices_sorted.data().get(), P);

	// hipEvent_t ev1, ev2, ev3;
	// hipEventCreate(&ev1);
	// hipEventCreate(&ev2);
	// hipEventCreate(&ev3);

	uint32_t num_boxes2 = (P + BOX_SIZE2 - 1) / BOX_SIZE2;
	thrust::device_vector<MinMax> boxes(num_boxes2);
	// hipEventRecord(ev1);
	boxMinMax<BOX_SIZE2> << <num_boxes2, BOX_SIZE2 >> > (P, points, indices_sorted.data().get(), boxes.data().get());
	// hipEventRecord(ev2);

	int num_blocks = (P + 255) / 256;
	boxKnn2 << <num_blocks, 256>> > (K, P, points, indices_sorted.data().get(), boxes.data().get(), dists, index_space);
	// hipEventRecord(ev3);

	// hipEventSynchronize(ev3);
	// float ms1, ms2;
	// hipEventElapsedTime(&ms1, ev1, ev2);
	// hipEventElapsedTime(&ms2, ev2, ev3);

	// std::cout << "First part: " << ms1 << std::endl;
	// std::cout << "Second part: " << ms2 << std::endl;

	hipFree(result);
}

__global__ void boxKnnQ(
	int K,
	uint32_t P,
	const float3* __restrict__ points,
	uint32_t Q,
	const int* __restrict__ query_indices,
	const bool* __restrict__ is_neighbor,
	const uint32_t* __restrict__ indices,
	const uint32_t* __restrict__ i2p,
	const MinMax* __restrict__ boxes,
	float* __restrict__ dists,
	int* __restrict__ index_space)
{
	int q_idx = cg::this_grid().thread_rank();
	if (q_idx >= Q)
		return;

	int i_idx = query_indices[q_idx];
	const float3 point = points[i_idx];

	float* best = dists + q_idx * K;
	int* best_ind = index_space + q_idx * K;
	for (int i = 0; i < K; i++)
		best[i] = FLT_MAX;
	float reject = FLT_MAX;

	int p_idx = i2p[i_idx];

	int b = p_idx / BOX_SIZE2;
	int lo = b, hi = b;

	const int num_boxes = (P + BOX_SIZE2 - 1) / BOX_SIZE2;

	for (int iter = 0; iter < num_boxes; iter++)
	{
		MinMax box = boxes[b];
		float dist = distBoxPoint(box, point);
		if (dist < reject)
		{
			for (int i = b * BOX_SIZE2; i < min(P, (b + 1) * BOX_SIZE2); i++)
			{
				if (i == p_idx)
					continue;
				const int other_idx = indices[i];

				if(is_neighbor[other_idx])
					updateKBest2(reject, K, other_idx, point, points[other_idx], best, best_ind);
			}
		}
		bool odd = iter & 1;
		b = (odd && hi == num_boxes - 1) || (!odd && lo > 0) ? --lo : ++hi;
	}
}

__global__ void fillIndex2Pos(int P, int N, int* neighbor_indices, bool* __restrict__ is_neighbor, uint32_t* __restrict__ ind_sorted, uint32_t* __restrict__ ind2pos)
{
	int idx = cg::this_grid().thread_rank();
	if (idx >= P)
		return;
	int ind = ind_sorted[idx];
	ind2pos[ind] = idx;

	if (idx < N)
	{
		is_neighbor[neighbor_indices[idx]] = true;
	}
}

void SimpleKNN::knn_indexQ(int K, int P, float3* points, int Q, int* query_indices, int N, int* neighbor_indices, float* dists, int* index_space)
{
	float3* result;
	hipMalloc(&result, sizeof(float3));
	size_t temp_storage_bytes;

	float3 init = { 0, 0, 0 }, minn, maxx;

	hipcub::DeviceReduce::Reduce(nullptr, temp_storage_bytes, points, result, P, CustomMin(), init);
	thrust::device_vector<char> temp_storage(temp_storage_bytes);

	hipcub::DeviceReduce::Reduce(temp_storage.data().get(), temp_storage_bytes, points, result, P, CustomMin(), init);
	hipMemcpy(&minn, result, sizeof(float3), hipMemcpyDeviceToHost);

	hipcub::DeviceReduce::Reduce(temp_storage.data().get(), temp_storage_bytes, points, result, P, CustomMax(), init);
	hipMemcpy(&maxx, result, sizeof(float3), hipMemcpyDeviceToHost);

	thrust::device_vector<uint32_t> morton(P);
	thrust::device_vector<uint32_t> morton_sorted(P);
	coord2Morton << <(P + 255) / 256, 256 >> > (P, points, minn, maxx, morton.data().get());

	thrust::device_vector<uint32_t> indices(P);
	thrust::sequence(indices.begin(), indices.end());
	thrust::device_vector<uint32_t> indices_sorted(P);

	hipcub::DeviceRadixSort::SortPairs(nullptr, temp_storage_bytes, morton.data().get(), morton_sorted.data().get(), indices.data().get(), indices_sorted.data().get(), P);
	temp_storage.resize(temp_storage_bytes);

	hipcub::DeviceRadixSort::SortPairs(temp_storage.data().get(), temp_storage_bytes, morton.data().get(), morton_sorted.data().get(), indices.data().get(), indices_sorted.data().get(), P);

	// hipEvent_t ev1, ev2, ev3;
	// hipEventCreate(&ev1);
	// hipEventCreate(&ev2);
	// hipEventCreate(&ev3);

	uint32_t num_boxes2 = (P + BOX_SIZE2 - 1) / BOX_SIZE2;
	thrust::device_vector<MinMax> boxes(num_boxes2);
	// hipEventRecord(ev1);
	boxMinMax<BOX_SIZE2> << <num_boxes2, BOX_SIZE2 >> > (P, points, indices_sorted.data().get(), boxes.data().get());
	// hipEventRecord(ev2);

	int num_blocks = (P + 255) / 256;
	thrust::device_vector<uint32_t> index2pos(P);
	thrust::device_vector<bool> is_neighbor(P, false);
	fillIndex2Pos << <num_blocks, 256 >> > (P, N, neighbor_indices, is_neighbor.data().get(), indices_sorted.data().get(), index2pos.data().get());

	int num_blocks2 = (Q + 255) / 256;
	boxKnnQ << <num_blocks2, 256 >> > (K, P, points, Q, query_indices, is_neighbor.data().get(), indices_sorted.data().get(), index2pos.data().get(), boxes.data().get(), dists, index_space);
	// hipEventRecord(ev3);

	// hipEventSynchronize(ev3);
	// float ms1, ms2;
	// hipEventElapsedTime(&ms1, ev1, ev2);
	// hipEventElapsedTime(&ms2, ev2, ev3);

	// std::cout << "First part: " << ms1 << std::endl;
	// std::cout << "Second part: " << ms2 << std::endl;

	hipFree(result);
}
